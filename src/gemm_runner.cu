#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <cassert>
#include <stdexcept>
#include <hip/hip_runtime.h>

#include "lib/test.hpp"
#include "lib/macros.cuh"
#include "kernels/gemm_seq.hpp"
#include "kernels/gemm_kernels.hpp"

int main() {
  CudaDeviceInfo();
  const int64_t float_calculation_num = 2*static_cast<uint64_t>(BatchSize)*Nnn*Nii;
  auto input_length = BatchSize*Nii;
  auto output_length = BatchSize*Nnn;
  auto weight_length = Nii * Nnn;
  auto gemm_test = Test<float, decltype(gemm_naive<BatchSize, Nii, Nnn, BatchSize, 16>)>
    (input_length, output_length, weight_length, float_calculation_num, "GEMM ", 10);
  gemm_test.run_seq(gemm_seq<BatchSize, Nii, Nnn>);
  gemm_test.test_cuda(gemm_naive<BatchSize, Nii, Nnn, BatchSize, 16>, "CUDA NAIVE");
  gemm_test.test_cuda(gemm_coalescing<BatchSize, Nii, Nnn, BatchSize, 16>, "CUDA coalescing");
  gemm_test.test_cuda(gemm_naive_shared<BatchSize, Nii, Nnn, BatchSize, 64, 512>, "CUDA NAIVE SHARED");
  gemm_test.test_cuda(gemm_shared<BatchSize, Nii, Nnn, BatchSize, 64, 64>, "CUDA SHARED");
  gemm_test.test_cuda(gemm_block_tiling<BatchSize, Nii, Nnn, BatchSize, 64, 64, 2, 2>, "CUDA TILING");
  gemm_test.test_cuda(gemm_vectorize<BatchSize, Nii, Nnn, BatchSize, 64, 64, 2, 2>, "CUDA VECTERIZE");
}
