#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <chrono>
#include <stdexcept>
#include <hip/hip_runtime.h>

#include "lib/utils.h"
#include "lib/macros.cuh"

using std::clog;
using std::endl;
using std::max;

#define input(ni, h, w) input[(ni)*NyPAD*NxPAD + (h)*NxPAD + (w)]
#define output(nn, h, w) output[(nn)*NySCL*NxSCL + (h)*NxSCL + (w)]
#define weight(nn, ni, p, q) weight[(nn)*Ni*Ky*Kx + (ni)*Ky*Kx + (p)*Kx + (q)]

// Sequential CNN implementation
// input: Ni * NyPAD * NxPAD
// weight: Nn * Ni * Ky * Kx
// output: Nn * NySCL * NxSCL
void ConvSequential(const float *input,
    const float *weight,
    float *output) {

  for(int nn = 0; nn < Nn; ++nn) {
    for(int ny = 0; ny < Ny; ny += Sy) {
      for(int nx = 0; nx < Nx; nx += Sx) {
        int xout = nx / Sx;
        int yout = ny / Sy;
        float sum = 0.0f;

        for(int ni = 0; ni < Ni; ++ni) {
          for(int ky = 0; ky < Ky; ++ky) {
            for(int kx = 0; kx < Kx; ++kx) {
              sum += weight(nn, ni, ky, kx) * input(ni, ny+ky, nx+kx);
            }
          }
        }

        // Perform Relu
        output(nn, yout, xout) = max(0.0f, sum);
      }
    }
  }
}

// kernal number that each thread need to deal with
constexpr int KERNAL_COUNT = (Nn / GRIDDIMZ / BLOCKSIZEZ);
constexpr int BLOCK_IN_X = (BLOCKSIZEX + Kx - 1);
constexpr int BLOCK_IN_Y = (BLOCKSIZEY + Ky - 1);
constexpr int KERNEL_SQUARE = Kx * Ky;

// Sequential CNN implementation
// input: Ni * NyPAD * NxPAD
// weight: Nn * Ni * Ky * Kx
// output: Nn * NySCL * NxSCL
__global__ void conv_gpu(float* input, float* weight, float* output) {
    __shared__ float weight_blocked[BLOCK_CHANNEL][Kx][Ky];
    __shared__ float input_blocked[BLOCK_CHANNEL][BLOCK_IN_X][BLOCK_IN_X];
    float output_thread[KERNAL_COUNT];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int kernalOffset = blockIdx.z * KERNAL_COUNT;

    int row = blockIdx.y * BLOCKSIZEY;
    int col = blockIdx.x * BLOCKSIZEX;

    // set bias
    #pragma unroll
    for (int k = kernalOffset; k < kernalOffset + KERNAL_COUNT; k++) {
        output_thread[k - kernalOffset] = 0.0f;
    }

    for(int ni = 0; ni < Nn; ni += BLOCK_CHANNEL) {

        /* Step1. load input to shared memory */
        int x, y;
        #pragma unroll
        for(int i = 0; i < BLOCK_CHANNEL; ++i) {
            #pragma unroll
            for (int offset = ty*BLOCKSIZEX + tx; offset < BLOCK_IN_X * BLOCK_IN_Y; offset += BLOCKSIZEX*BLOCKSIZEY) {
                x = offset % BLOCK_IN_X;
                y = offset / BLOCK_IN_X;
                input_blocked[i][y][x] = input(ni + i, row + y, col + x);
            }
        }

        for (int k = kernalOffset; k < kernalOffset + KERNAL_COUNT; k++) {
            /* Step2. load weight to shared memory */
            #pragma unroll
            for(int offset = ty*BLOCKSIZEX + tx; offset < KERNEL_SQUARE * BLOCK_CHANNEL; offset += BLOCKSIZEX*BLOCKSIZEY) {
                int cz = offset / KERNEL_SQUARE;
                int cz_off = offset % KERNEL_SQUARE;
                int cx = cz_off % Kx;
                int cy = cz_off / Kx;
                weight_blocked[cz][cy][cx] = weight(k, ni + cz, cy, cx);
            }
            __syncthreads();

            /* Step3. Computation */
            for (int kk = 0; kk < BLOCK_CHANNEL; kk++) {
                for (int ky = 0; ky < Ky; ky++) {
                    for (int kx = 0; kx < Kx; kx++) {
                        output_thread[k - kernalOffset] += input_blocked[kk][ty+ky][tx+kx] * weight_blocked[kk][ky][kx];
                    }
                }
            }
            __syncthreads();
        }
    }

    // Relu
    for (int k = 0; k < KERNAL_COUNT; k++) {
        output(kernalOffset + k, row + ty, col + tx) = max(0.0f, output_thread[k]);
    }
}

int main() {
  auto input_length = Ni * NyPAD * NxPAD; auto input_size = input_length * sizeof(float);
  auto output_length = Nn * NySCL * NxSCL; auto output_size = output_length * sizeof(float);
  auto weight_length = Nn * Ni * Ky * Kx; auto weight_size = weight_length * sizeof(float);
  float* input = static_cast<float*>(malloc(input_size));
  float* output = static_cast<float*>(malloc(output_size));
  float* weight = static_cast<float*>(malloc(weight_size));
  auto sta = std::chrono::steady_clock::now();
  GenerateRandomMatrix(input, input_length);
  GenerateRandomMatrix(weight, weight_length);
  std::chrono::duration<double> rand_duration = std::chrono::steady_clock::now() - sta;
  clog << "[Generate Random Matrix]\tTimeCost:" << rand_duration.count() << "ns" << std::endl;

  sta = std::chrono::steady_clock::now();
  ConvSequential(input, weight, output);
  std::chrono::duration<double> conv_seq_duration = std::chrono::steady_clock::now() - sta;
  clog << "[Conv Sequence]\tTimeCost:" << conv_seq_duration.count() << "ns" << std::endl;

  float* cuda_output = static_cast<float*>(malloc(output_size));
  float* g_input, *g_weight, *g_output;
  hipMalloc((float**)&g_input, input_size);
  hipMalloc((float**)&g_weight, weight_size);
  hipMalloc((float**)&g_output, output_size);
  hipMemcpy(g_input, input, input_size, hipMemcpyHostToDevice);
  hipMemcpy(g_weight, weight, weight_size, hipMemcpyHostToDevice);

  constexpr int GRIDDIMX = (Nx / BLOCKSIZEX);
  constexpr int GRIDDIMY = (Ny / BLOCKSIZEY);
  auto block = dim3(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
  auto grid = dim3(GRIDDIMX, GRIDDIMY, GRIDDIMZ);
  std::clog << "Using thread block dims: " << block.x << ' ' << block.y << ' ' << block.z << '\n';
  std::clog << "Using grid dims: " << grid.x << ' ' << grid.y << ' ' << grid.z << '\n';
  hipSetDevice(0);

  sta = std::chrono::steady_clock::now();
  conv_gpu<<<grid, block>>>(g_input, g_weight, g_output);
  CUDA_CHECK(hipDeviceSynchronize());
  std::chrono::duration<double> conv_gpu_duration = std::chrono::steady_clock::now() - sta;
  clog << "[Conv CUDA]\tTimeCost:" << conv_gpu_duration.count() << "ns" << std::endl;

  hipMemcpy(cuda_output, g_output, output_size, hipMemcpyDeviceToHost);
  if (IsDiffMatrix(cuda_output, output, output_length)) {
    clog << "FAIL" << endl;
  } else {
    clog << "PASS" << endl;
  }
}
